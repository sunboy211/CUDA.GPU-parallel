#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdint.h>
#include<iostream>
#include<string.h>
#include<ctype.h>
#include<stdlib.h>
#include<>

typedef unsigned int ui;
typedef unsigned long ul;
typedef unsigned char uch;

uch *TheImg,*CopyImg;
uch *GPUImg,*GPUCopyImg,*GPUResult;

//hipGetDeviceCount() 一共几个GPU
//hipSetDevice() 选择哪个GPU
//hipGetDeviceProperties GPU参数
//GPUProp.maxGridSize[0],GPUProp.maxGridSize[1],GPUProp.maxGridSize[2]分别表示x,y,z维度上可启动的最大线程块数

int main(int argc,char** argv){
  hipError_t cudaStatus,cudaStatus2;
  hipDeviceProp_t GPUprop;
  ul SupportedKBlocks,SupportedMBlocks,MaxThrPerBlk;
  char SupportedBlocks[100];

  int NumGPUs=0; hipGetDeviceCount(&NumGPUs);
  if(NumGPUs==0){
    printf("\nNo cuda device is available.\n"); exit(EXIT_FAILURE);
  }
  cudaStatus=NumGPUs(0)
}
