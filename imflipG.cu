#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdint.h>
#include<iostream>
#include<string.h>
#include<ctype.h>
#include<stdlib.h>
#include<>

typedef unsigned int ui;
typedef unsigned long ul;
typedef unsigned char uch;

uch *TheImg,*CopyImg;
uch *GPUImg,*GPUCopyImg,*GPUResult;

//hipGetDeviceCount() 一共几个GPU
//hipSetDevice() 选择哪个GPU
//hipGetDeviceProperties GPU参数
//GPUProp.maxGridSize[0],GPUProp.maxGridSize[1],GPUProp.maxGridSize[2]分别表示x,y,z维度上可启动的最大线程块数

//下面是Nvidia API初始化并查询GPU
int main(int argc,char** argv){
  hipError_t cudaStatus,cudaStatus2;
  hipDeviceProp_t GPUprop;
  ul SupportedKBlocks,SupportedMBlocks,MaxThrPerBlk;
  char SupportedBlocks[100];

  int NumGPUs=0; hipGetDeviceCount(&NumGPUs);
  if(NumGPUs==0){
    printf("\n No cuda device is available.\n"); exit(EXIT_FAILURE);
  }
  cudaStatus=hipSetDevice(0);
  if(cudaStatus!=hipSuccess){
    fprintf(stderr,"cudasetDevice failed!No cuda-capable GPU is installed?");
    exit(EXIT_FAILURE);
  }
  hipGetDeviceProperties(&GPUprop,0);
  SupportedKBlocks=(ui)GPUProp.maxGridSize[0] * (ui)GPUProp.maxGridSize[1] * (ui)GPUProp.maxGridSize[2] / 1024;
  SupportedMBlocks=SupportedKBlocks / 1024;
  MaxThrPerBlk=(ui)GPUProp.maxThreadsPerBlock;
}
